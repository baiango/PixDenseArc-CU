#include "hip/hip_runtime.h"
#include "ycocgp.cuh"

__host__ __device__
YCoCgp pda_new_ycocgp() {
	return YCoCgp{64.0f, 0.0f, 0.0f};
}

__host__ __device__
YCoCgp pda_new_black_ycocgp() {
	return YCoCgp{0.0f, 0.0f, 0.0f};
}

__host__ __device__
YCoCgp pda_rgb_to_ycocgp(RGB888 rgb) {
	f32 y = 0.5 * rgb.r + rgb.g + 0.5000000001f * rgb.b;
	f32 co = rgb.r + -1.000000001f * rgb.b;
	f32 cg = -0.5 * rgb.r + rgb.g + -0.5000000001f * rgb.b;
	return YCoCgp{y, co, cg};
}

__host__ __device__
RGB888 pda_ycocgp_to_rgb(YCoCgp ycocgp) {
	u8 r = (ycocgp.y + ycocgp.co - ycocgp.cg) / 2.0f;
	u8 g = (ycocgp.y + ycocgp.cg) / 2.0f;
	u8 b = (ycocgp.y - ycocgp.co - ycocgp.cg) / 2.0f;
	return RGB888{r,g,b};
}
/* ----- �nit ----- */
MunitTest ycocgp_tests[] = {
	{"RGB to YCoCg+", pda_test_rgb_to_ycocgp, NULL, NULL, MUNIT_TEST_OPTION_NONE, NULL},
	{"Are equal YCoCg+", pda_test_are_equal_ycocgp, NULL, NULL, MUNIT_TEST_OPTION_NONE, NULL},
	{NULL, NULL, NULL, NULL, MUNIT_TEST_OPTION_NONE, NULL },
};

MunitSuite ycocgp_suite = {
	"/YCoCg+/", /* name */
	ycocgp_tests, /* tests */
	NULL, /* suites */
	1, /* iterations */
	MUNIT_SUITE_OPTION_NONE /* options */
};

u1 pda_are_equal_ycocgp(YCoCgp a, YCoCgp b) {
	f32 EPSILON = 1.0e-5;
	return (fabs(a.y - b.y) < EPSILON) &&
		(fabs(a.co - b.co) < EPSILON) &&
		(fabs(a.cg - b.cg) < EPSILON);
}

MunitResult pda_test_are_equal_ycocgp(const MunitParameter params[], void* user_data) {
	return pda_are_equal_ycocgp({0.1f, 0.1f, 0.1f}, {0.1f, 0.1f, 0.1f}) ? MUNIT_OK : MUNIT_FAIL;
}

MunitResult pda_test_rgb_to_ycocgp(const MunitParameter params[], void* user_data) {
	YCoCgp col1 = pda_rgb_to_ycocgp(RGB888{0x20, 0x20, 0x20});
	YCoCgp col2 = {64.0f, 0.0f, 0.0f};
	return pda_are_equal_ycocgp(col1, col2) ? MUNIT_OK : MUNIT_FAIL;
}

MunitResult pda_test_ycocgp_to_rgb(const MunitParameter params[], void* user_data) {
	return pda_are_equal_rgb888(pda_ycocgp_to_rgb({15.0f, -30.0f, -15.0f}), RGB888{0, 0, 30}) ? MUNIT_OK : MUNIT_FAIL;
}
