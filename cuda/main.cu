﻿#include "hip/hip_runtime.h"
#include ""
#include "rgb.cuh"
#include "ycocgp.cuh"

int main() {
	munit_suite_main(&rgb_suite, NULL, NULL, NULL);
	munit_suite_main(&ycocgp_suite, NULL, NULL, NULL);
	return 0;
}
