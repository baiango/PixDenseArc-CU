#include "rgb.cuh"

__host__ __device__
u1 pda_are_equal_rgb888(RGB888 a, RGB888 b) {
	return (a.r == b.r) &&
		(a.g == b.g) &&
		(a.b == b.b);
}

/* ----- �nit ----- */
MunitTest rgb_tests[] = {
	{"Are equal RGB888", pda_test_are_equal_rgb888, NULL, NULL, MUNIT_TEST_OPTION_NONE, NULL},
	{NULL, NULL, NULL, NULL, MUNIT_TEST_OPTION_NONE, NULL },
};

MunitSuite rgb_suite = {
	"/RGB/", /* name */
	rgb_tests, /* tests */
	NULL, /* suites */
	1, /* iterations */
	MUNIT_SUITE_OPTION_NONE /* options */
};

MunitResult pda_test_are_equal_rgb888(const MunitParameter params[], void* user_data) {
	return pda_are_equal_rgb888({200, 200, 200}, {200, 200, 200}) ? MUNIT_OK : MUNIT_FAIL;
}
